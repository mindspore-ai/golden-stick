#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "fake_quant_impl.cuh"

/**
 * Calculate fake quant output according by nudge min, nudge max, nudge scale.
 * @param input - array
 * @param output - array
 * @param total_size - int, purpose for cal the per channel number in filters
 * @param channel_size - int, purpose for cal the per channel number in filters
 * @param nudge_min - array
 * @param nudge_max - array
 * @param scale - array
 * @return
 */
__global__ void FakeQuantPerChannel(const float *input, float *output, const int total_size, const int channel_size,
                                    const float *nudge_min, const float *nudge_max, const float *scale) {
  float input_x = 0.f;
  int nudge_input = 0;
  int channel_idx = 0;
  int per_channel_num = total_size / channel_size;

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < total_size; i += blockDim.x * gridDim.x) {
    input_x = input[i];
    channel_idx = floor(static_cast<double>(i) / static_cast<double>(per_channel_num));
    // clamp input x
    if (input_x < nudge_min[channel_idx]) {
      input_x = nudge_min[channel_idx];
    }
    if (input_x > nudge_max[channel_idx]) {
      input_x = nudge_max[channel_idx];
    }
    // clamp shift
    nudge_input = floor((input_x - nudge_min[channel_idx]) / scale[channel_idx] + 0.5f);

    // quantize
    output[i] = nudge_input * scale[channel_idx] + nudge_min[channel_idx];
  }
}

void CalFakeQuantPerChannel(const float *input, float *output, const int total_size, const int channel_size,
                            const float *nudge_min, const float *nudge_max, const float *scale,
                            hipStream_t hip_stream) {
  FakeQuantPerChannel<<<GET_BLOCKS(total_size), GET_THREADS, 0, hip_stream>>>(input, output, total_size, channel_size,
                                                                               nudge_min, nudge_max, scale);
}

class FQPerChannelKernelAttr : public AotKernelData {
 public:
  int num_bits;
  bool training;
  bool symmetric;
  bool narrow_range;
  int quant_delay;
};

extern "C" int CustomFakeQuantPerChannelInit(int *ndims, int64_t **shapes, const char **dtypes, AotExtra *extra) {
  size_t num_channels = static_cast<size_t>(shapes[0][0]);
  extra->SetWorkSpace({num_channels * sizeof(float), num_channels * sizeof(float), num_channels * sizeof(float)});

  FQPerChannelKernelAttr *kernel_ptr = new FQPerChannelKernelAttr;
  kernel_ptr->num_bits = static_cast<int>(extra->Attr<int64_t>("num_bits"));
  kernel_ptr->training = extra->Attr<bool>("training");
  kernel_ptr->symmetric = extra->Attr<bool>("symmetric");
  kernel_ptr->narrow_range = extra->Attr<bool>("narrow_range");
  kernel_ptr->quant_delay = static_cast<int>(extra->Attr<int64_t>("quant_delay"));
  extra->SetKernelData(kernel_ptr);

  return 0;
}

int global_step = 0;

extern "C" int CustomFakeQuantPerChannel(int nparam, void **params, int *ndims, int64_t **shapes, const char **dtypes,
                                         void *stream, void *extra_void) {
  constexpr int TOTAL_PARAM_NUM = 3 + 1 + 3;  // input 3, output 1, workspace 3
  constexpr int IO_NUM = 4;
  constexpr int OUTPUT_INDEX = 3;

  if (nparam != TOTAL_PARAM_NUM) {
    return 1;
  }

  for (int index = 0; index < IO_NUM; index++) {
    if (strcmp(dtypes[index], "float32") != 0) {
      return 2;
    }
  }

  float *input_data = static_cast<float *>(params[0]);
  float *input_min = static_cast<float *>(params[1]);
  float *input_max = static_cast<float *>(params[2]);
  float *output = static_cast<float *>(params[3]);
  float *w_scale = static_cast<float *>(params[4]);
  float *w_nudge_min = static_cast<float *>(params[5]);
  float *w_nudge_max = static_cast<float *>(params[6]);

  int size = 1;
  for (int i = 0; i < ndims[OUTPUT_INDEX]; i++) {
    size *= shapes[OUTPUT_INDEX][i];
  }

  AotExtra *extra = static_cast<AotExtra *>(extra_void);
  auto kernel_ptr = static_cast<FQPerChannelKernelAttr *>(extra->KernelData());
  int num_bits = kernel_ptr->num_bits;
  if (num_bits <= 2 || num_bits >= 16) {
    return 3;
  }
  bool training = kernel_ptr->training;
  bool symmetric = kernel_ptr->symmetric;
  bool narrow_range = kernel_ptr->narrow_range;
  int quant_delay = kernel_ptr->quant_delay;
  if (quant_delay < 0) {
    return 3;
  }

  float quant_min = 0;
  float quant_max = (1 << num_bits) - 1;
  if (narrow_range) {
    quant_min++;
  }
  size_t num_channels = static_cast<size_t>(shapes[0][0]);

  if (training) {
    if (global_step >= quant_delay) {
      CalNudgePerChannel(input_min, input_max, quant_min, quant_max, w_nudge_min, w_nudge_max, w_scale, num_channels,
                         symmetric, reinterpret_cast<hipStream_t>(stream));
      CalFakeQuantPerChannel(input_data, output, size, num_channels, w_nudge_min, w_nudge_max, w_scale,
                             reinterpret_cast<hipStream_t>(stream));
    } else {
      CHECK_CUDA_RET_WITH_ERROR("FakeQuantPerChannel",
                                hipMemcpyAsync(output, input_data, size * sizeof(float), hipMemcpyDeviceToDevice,
                                                reinterpret_cast<hipStream_t>(stream)),
                                "Copy gpu memory failed.");
    }
    global_step++;
  } else {
    CalNudgePerChannel(input_min, input_max, quant_min, quant_max, w_nudge_min, w_nudge_max, w_scale, num_channels,
                       symmetric, reinterpret_cast<hipStream_t>(stream));
    CalFakeQuantPerChannel(input_data, output, size, num_channels, w_nudge_min, w_nudge_max, w_scale,
                           reinterpret_cast<hipStream_t>(stream));
  }

  return 0;
}
